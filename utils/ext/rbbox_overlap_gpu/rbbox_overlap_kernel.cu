#include "hip/hip_runtime.h"

#include "rbbox_overlap.hpp"
#include <vector>
#include <iostream>
#include <cmath>

#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      std::cout << hipGetErrorString(error) << std::endl; \
    } \
  } while (0)

#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))
int const threadsPerBlock = sizeof(unsigned long long) * 8;

__device__ inline float trangle_area(float * a, float * b, float * c) {
  return ((a[0] - c[0]) * (b[1] - c[1]) - (a[1] - c[1]) * (b[0] - c[0]))/2.0;
}

__device__ inline float area(float * int_pts, int num_of_inter) {

  float area = 0.0;
  for(int i = 0;i < num_of_inter - 2;i++) {
    area += fabs(trangle_area(int_pts, int_pts + 2 * i + 2, int_pts + 2 * i + 4));
  }
  return area;
}

__device__ inline void reorder_pts(float * int_pts, int num_of_inter) {



  if(num_of_inter > 0) {
    
    float center[2];
    
    center[0] = 0.0;
    center[1] = 0.0;

    for(int i = 0;i < num_of_inter;i++) {
      center[0] += int_pts[2 * i];
      center[1] += int_pts[2 * i + 1];
    }
    center[0] /= num_of_inter;
    center[1] /= num_of_inter;

    float vs[16];
    float v[2];
    float d;
    for(int i = 0;i < num_of_inter;i++) {
      v[0] = int_pts[2 * i]-center[0];
      v[1] = int_pts[2 * i + 1]-center[1];
      d = sqrt(v[0] * v[0] + v[1] * v[1]);
      v[0] = v[0] / d;
      v[1] = v[1] / d;
      if(v[1] < 0) {
        v[0]= - 2 - v[0];
      }
      vs[i] = v[0];
    }
    
    float temp,tx,ty;
    int j;
    for(int i=1;i<num_of_inter;++i){
      if(vs[i-1]>vs[i]){
        temp = vs[i];
        tx = int_pts[2*i];
        ty = int_pts[2*i+1];
        j=i;
        while(j>0&&vs[j-1]>temp){
          vs[j] = vs[j-1];
          int_pts[j*2] = int_pts[j*2-2];
          int_pts[j*2+1] = int_pts[j*2-1];
          j--;
        }
        vs[j] = temp;
        int_pts[j*2] = tx;
        int_pts[j*2+1] = ty;
      }
    }
  }

}
__device__ inline bool inter2line(float * pts1, float *pts2, int i, int j, float * temp_pts) {

  float a[2];
  float b[2];
  float c[2];
  float d[2];

  float area_abc, area_abd, area_cda, area_cdb;

  a[0] = pts1[2 * i];
  a[1] = pts1[2 * i + 1];

  b[0] = pts1[2 * ((i + 1) % 4)];
  b[1] = pts1[2 * ((i + 1) % 4) + 1];

  c[0] = pts2[2 * j];
  c[1] = pts2[2 * j + 1];

  d[0] = pts2[2 * ((j + 1) % 4)];
  d[1] = pts2[2 * ((j + 1) % 4) + 1];

  area_abc = trangle_area(a, b, c);
  area_abd = trangle_area(a, b, d);
  
  if(area_abc * area_abd >= -1e-5) {
    return false;
  }
  
  area_cda = trangle_area(c, d, a); 
  area_cdb = area_cda + area_abc - area_abd;

  if (area_cda * area_cdb >= -1e-5) {
    return false;
  }
  float t = area_cda / (area_abd - area_abc + 1e-6);      
    
  float dx = t * (b[0] - a[0]);
  float dy = t * (b[1] - a[1]);
  temp_pts[0] = a[0] + dx;
  temp_pts[1] = a[1] + dy;

  return true;
}

__device__ inline bool in_rect(float pt_x, float pt_y, float * pts) {
  
  double ab[2];
  double ad[2];
  double ap[2];

  double abab;
  double abap;
  double adad;
  double adap;

  ab[0] = pts[2] - pts[0];
  ab[1] = pts[3] - pts[1];

  ad[0] = pts[6] - pts[0];
  ad[1] = pts[7] - pts[1];

  ap[0] = pt_x - pts[0];
  ap[1] = pt_y - pts[1];

  abab = ab[0] * ab[0] + ab[1] * ab[1];
  abap = ab[0] * ap[0] + ab[1] * ap[1];
  adad = ad[0] * ad[0] + ad[1] * ad[1];
  adap = ad[0] * ap[0] + ad[1] * ap[1];
  bool result = (abab - abap >=  -1) and (abap >= -1) and (adad - adap >= -1) and (adap >= -1);
  return result;
}

__device__ inline int inter_pts(float * pts1, float * pts2, float * int_pts) {

  int num_of_inter = 0;

  for(int i = 0;i < 4;i++) {
    if(in_rect(pts1[2 * i], pts1[2 * i + 1], pts2)) {
      int_pts[num_of_inter * 2] = pts1[2 * i];
      int_pts[num_of_inter * 2 + 1] = pts1[2 * i + 1];
      num_of_inter++;
    }
     if(in_rect(pts2[2 * i], pts2[2 * i + 1], pts1)) {
      int_pts[num_of_inter * 2] = pts2[2 * i];
      int_pts[num_of_inter * 2 + 1] = pts2[2 * i + 1];
      num_of_inter++;
    }   
  }

  float temp_pts[2];

  for(int i = 0;i < 4;i++) {
    for(int j = 0;j < 4;j++) {
      bool has_pts = inter2line(pts1, pts2, i, j, temp_pts);
      if(has_pts) {
        int_pts[num_of_inter * 2] = temp_pts[0];
        int_pts[num_of_inter * 2 + 1] = temp_pts[1];
        num_of_inter++;
      }
    }
  }
  

  return num_of_inter;
}

__device__ inline void convert_region(float * pts , float const * const region) {

  float angle = region[4];
  float a_cos = cos(angle/180.0*3.1415926535);
  float a_sin = sin(angle/180.0*3.1415926535);

  float ctr_x = region[0];
  float ctr_y = region[1];

  float w = region[2];
  float h = region[3];

  float pts_x[4];
  float pts_y[4];

  pts_x[0] = - w / 2;
  pts_x[1] = w / 2;
  pts_x[2] = w / 2;
  pts_x[3] = - w / 2;

  pts_y[0] = - h / 2;
  pts_y[1] = - h / 2;
  pts_y[2] = h / 2;
  pts_y[3] = h / 2;

  for(int i = 0;i < 4;i++) {
    pts[7 - 2 * i - 1] = a_cos * pts_x[i] - a_sin * pts_y[i] + ctr_x;
    pts[7 - 2 * i] = a_sin * pts_x[i] + a_cos * pts_y[i] + ctr_y;
   
  }

}


__device__ inline float inter(float const * const region1, float const * const region2) {

  float pts1[8];
  float pts2[8];
  float int_pts[16];
  int num_of_inter;

  convert_region(pts1, region1);
  convert_region(pts2, region2);

  num_of_inter = inter_pts(pts1, pts2, int_pts);

  reorder_pts(int_pts, num_of_inter);

  return area(int_pts, num_of_inter);
  
  
}

__device__ inline float devRotateIoU(float const * const region1, float const * const region2) {

  if((fabs(region1[0] - region2[0]) < 1e-5) && (fabs(region1[1] - region2[1]) < 1e-5) && (fabs(region1[2] - region2[2]) < 1e-5) && (fabs(region1[3] - region2[3]) < 1e-5) && (fabs(region1[4] - region2[4]) < 1e-5)) {
    return 1.0;
  }

  float area1 = region1[2] * region1[3];
  float area2 = region2[2] * region2[3];
  float area_inter = inter(region1, region2);

  if (region1[2] < 0.1 | region1[3] < 0.1 | region2[2] < 0.1 | region2[3] < 0.1){
    area_inter = 0;
  }

  float result = area_inter / (area1 + area2 - area_inter + 1e-6);

  if(result < 0 | result > 1) {
    result = 0.;
  }
  return result;
  
  
}

__global__ void overlaps_kernel(const int N, const int K, const float* dev_boxes,
                           const float * dev_query_boxes, float* dev_overlaps) {

  const int col_start = blockIdx.y;
  const int row_start = blockIdx.x;

  const int row_size =
        min(N - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(K - col_start * threadsPerBlock, threadsPerBlock);


  __shared__ float block_boxes[threadsPerBlock * 5];
  __shared__ float block_query_boxes[threadsPerBlock * 5];
  if (threadIdx.x < col_size) {
    block_query_boxes[threadIdx.x * 5 + 0] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
    block_query_boxes[threadIdx.x * 5 + 1] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
    block_query_boxes[threadIdx.x * 5 + 2] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
    block_query_boxes[threadIdx.x * 5 + 3] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
    block_query_boxes[threadIdx.x * 5 + 4] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }

  if (threadIdx.x < row_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 5 + 4];
  }

  __syncthreads();

  if (threadIdx.x < row_size) {

    for(int i = 0;i < col_size; i++) {
      int offset = row_start*threadsPerBlock * K + col_start*threadsPerBlock + threadIdx.x*K+ i ;
      dev_overlaps[offset] = devRotateIoU(block_boxes + threadIdx.x * 5, block_query_boxes + i * 5);
    }

  }
}


__global__ void rotate_nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * 6];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 6 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 6 + 0];
    block_boxes[threadIdx.x * 6 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 6 + 1];
    block_boxes[threadIdx.x * 6 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 6 + 2];
    block_boxes[threadIdx.x * 6 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 6 + 3];
    block_boxes[threadIdx.x * 6 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 6 + 4];
    block_boxes[threadIdx.x * 6 + 5] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 6 + 5];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * 6;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devRotateIoU(cur_box, block_boxes + i * 6) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}


void _set_device(int device_id) {
  int current_device;
  CUDA_CHECK(hipGetDevice(&current_device));
  if (current_device == device_id) {
    return;
  }
  // The call to hipSetDevice must come before any calls to Get, which
  // may perform initialization using the GPU.
  CUDA_CHECK(hipSetDevice(device_id));
}


void _overlaps(float* overlaps,const float* boxes,const float* query_boxes, int n, int k, int device_id) {

  _set_device(device_id);

  float* overlaps_dev = NULL;
  float* boxes_dev = NULL;
  float* query_boxes_dev = NULL;


  CUDA_CHECK(hipMalloc(&boxes_dev,
                        n * 5 * sizeof(float)));

  CUDA_CHECK(hipMemcpy(boxes_dev,
                        boxes,
                        n * 5 * sizeof(float),
                        hipMemcpyHostToDevice));

  CUDA_CHECK(hipMalloc(&query_boxes_dev,
                        k * 5 * sizeof(float)));

  CUDA_CHECK(hipMemcpy(query_boxes_dev,
                        query_boxes,
                        k * 5 * sizeof(float),
                        hipMemcpyHostToDevice));

  CUDA_CHECK(hipMalloc(&overlaps_dev,
                        n * k * sizeof(float)));

  dim3 blocks(DIVUP(n, threadsPerBlock),
              DIVUP(k, threadsPerBlock));

  dim3 threads(threadsPerBlock);

  overlaps_kernel<<<blocks, threads>>>(n, k,
                                    boxes_dev,
                                    query_boxes_dev,
                                    overlaps_dev);

  CUDA_CHECK(hipMemcpy(overlaps,
                        overlaps_dev,
                        n * k * sizeof(float),
                        hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(overlaps_dev));
  CUDA_CHECK(hipFree(boxes_dev));
  CUDA_CHECK(hipFree(query_boxes_dev));

}


void _rotate_nms(int* keep_out, int* num_out, const float* boxes_host, int boxes_num,
          int boxes_dim, float nms_overlap_thresh, int device_id) {
  _set_device(device_id);

  float* boxes_dev = NULL;
  unsigned long long* mask_dev = NULL;

  const int col_blocks = DIVUP(boxes_num, threadsPerBlock);

  CUDA_CHECK(hipMalloc(&boxes_dev,
                        boxes_num * boxes_dim * sizeof(float)));
  CUDA_CHECK(hipMemcpy(boxes_dev,
                        boxes_host,
                        boxes_num * boxes_dim * sizeof(float),
                        hipMemcpyHostToDevice));

  CUDA_CHECK(hipMalloc(&mask_dev,
                        boxes_num * col_blocks * sizeof(unsigned long long)));

  dim3 blocks(DIVUP(boxes_num, threadsPerBlock),
              DIVUP(boxes_num, threadsPerBlock));
  dim3 threads(threadsPerBlock);
  rotate_nms_kernel<<<blocks, threads>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_dev,
                                  mask_dev);

  std::vector<unsigned long long> mask_host(boxes_num * col_blocks);
  CUDA_CHECK(hipMemcpy(&mask_host[0],
                        mask_dev,
                        sizeof(unsigned long long) * boxes_num * col_blocks,
                        hipMemcpyDeviceToHost));

  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[num_to_keep++] = i;
      unsigned long long *p = &mask_host[0] + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }
  *num_out = num_to_keep;

  CUDA_CHECK(hipFree(boxes_dev));
  CUDA_CHECK(hipFree(mask_dev));
}
